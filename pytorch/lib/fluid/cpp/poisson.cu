#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>
#include <hipsparse.h>
#include <vector>
#include <cassert>



int main() {
    // --- create library handles:
    hipsolverSpHandle_t cusolver_handle;
    hipsolverStatus_t cusolver_status;
    cusolver_status = hipsolverSpCreate(&cusolver_handle);
    std::cout << "status create cusolver handle: " << cusolver_status << std::endl;

    hipsparseHandle_t cusparse_handle;
    hipsparseStatus_t cusparse_status;
    cusparse_status = hipsparseCreate(&cusparse_handle);
    std::cout << "status create cusparse handle: " << cusparse_status << std::endl;

    // --- prepare matrix:
    int Nrows = 4;
    int Ncols = 4;
    std::vector<float> csrVal;
    std::vector<int> cooRow;
    std::vector<int> csrColInd;
    std::vector<float> b;

    assemble_poisson_matrix_coo(csrVal, cooRow, csrColInd, b, Nrows, Ncols);

    int nnz = csrVal.size();
    int m = Nrows * Ncols;
    std::vector<int> csrRowPtr(m+1);

    // --- prepare solving and copy to GPU:
    std::vector<float> x(m);
    float tol = 1e-5;
    int reorder = 0;
    int singularity = 0;

    float *db, *dcsrVal, *dx;
    int *dcsrColInd, *dcsrRowPtr, *dcooRow;
    hipMalloc((void**)&db, m*sizeof(float));
    hipMalloc((void**)&dx, m*sizeof(float));
    hipMalloc((void**)&dcsrVal, nnz*sizeof(float));
    hipMalloc((void**)&dcsrColInd, nnz*sizeof(int));
    hipMalloc((void**)&dcsrRowPtr, (m+1)*sizeof(int));
    hipMalloc((void**)&dcooRow, nnz*sizeof(int));

    hipMemcpy(db, b.data(), b.size()*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dcsrVal, csrVal.data(), csrVal.size()*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dcsrColInd, csrColInd.data(), csrColInd.size()*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dcooRow, cooRow.data(), cooRow.size()*sizeof(int), hipMemcpyHostToDevice);

    cusparse_status = hipsparseXcoo2csr(cusparse_handle, dcooRow, nnz, m,
                                       dcsrRowPtr, HIPSPARSE_INDEX_BASE_ZERO);
    std::cout << "status cusparse coo2csr conversion: " << cusparse_status << std::endl;

    hipDeviceSynchronize(); // matrix format conversion has to be finished!

    // --- everything ready for computation:

    hipsparseMatDescr_t descrA;

    cusparse_status = hipsparseCreateMatDescr(&descrA);
    std::cout << "status cusparse createMatDescr: " << cusparse_status << std::endl;

    // optional: print dense matrix that has been allocated on GPU

    std::vector<float> A(m*m, 0);
    float *dA;
    hipMalloc((void**)&dA, A.size()*sizeof(float));

    hipsparseScsr2dense(cusparse_handle, m, m, descrA, dcsrVal,
                       dcsrRowPtr, dcsrColInd, dA, m);

    hipMemcpy(A.data(), dA, A.size()*sizeof(float), hipMemcpyDeviceToHost);
    std::cout << "A: \n";
    for (int i = 0; i < m; ++i) {
        for (int j = 0; j < m; ++j) {
            std::cout << A[i*m + j] << " ";
        }
        std::cout << std::endl;
    }

    hipFree(dA);

    std::cout << "b: \n";
    hipMemcpy(b.data(), db, (m)*sizeof(int), hipMemcpyDeviceToHost);
    for (auto a : b) {
        std::cout << a << ",";
    }
    std::cout << std::endl;


    // --- solving!!!!

//    cusolver_status = hipsolverSpScsrlsvchol(cusolver_handle, m, nnz, descrA, dcsrVal,
//                       dcsrRowPtr, dcsrColInd, db, tol, reorder, dx,
//                       &singularity);

     cusolver_status = hipsolverSpScsrlsvqr(cusolver_handle, m, nnz, descrA, dcsrVal,
                        dcsrRowPtr, dcsrColInd, db, tol, reorder, dx,
                        &singularity);

    hipDeviceSynchronize();

    std::cout << "singularity (should be -1): " << singularity << std::endl;

    std::cout << "status cusolver solving (!): " << cusolver_status << std::endl;

    hipMemcpy(x.data(), dx, m*sizeof(float), hipMemcpyDeviceToHost);

    // relocated these 2 lines from above to solve (2):
    cusparse_status = hipsparseDestroy(cusparse_handle);
    std::cout << "status destroy cusparse handle: " << cusparse_status << std::endl;

    cusolver_status = hipsolverSpDestroy(cusolver_handle);
    std::cout << "status destroy cusolver handle: " << cusolver_status << std::endl;

    for (auto a : x) {
        std::cout << a << " ";
    }
    std::cout << std::endl;



    hipFree(db);
    hipFree(dx);
    hipFree(dcsrVal);
    hipFree(dcsrColInd);
    hipFree(dcsrRowPtr);
    hipFree(dcooRow);

    return 0;
}

